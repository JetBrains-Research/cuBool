/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/cuda_instance.hpp>
#include <core/error.hpp>
#include <string>
#include <cassert>
#include <cstring>

namespace cubool {

    volatile CudaInstance* CudaInstance::gInstance = nullptr;

    CudaInstance::CudaInstance(bool useManagedMemory) {
        gInstance = this;
        mMemoryType = useManagedMemory? Managed: Default;
    }

    CudaInstance::~CudaInstance() {
        assert(mHostAllocCount == 0);
        assert(mDeviceAllocCount == 0);

        gInstance = nullptr;
    }

    void CudaInstance::allocateOnGpu(void* &ptr, size_t size) const {
        hipError_t error;

        switch (mMemoryType) {
            case MemType::Default:
                error = hipMalloc(&ptr, size);
                break;
            case MemType::Managed:
                error = hipMallocManaged(&ptr, size, hipMemAttachGlobal);
                break;
            default:
                RAISE_ERROR(MemOpFailed, "Failed to fined suitable allocator");
        }

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to allocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount++;
    }

    void CudaInstance::deallocateOnGpu(void* ptr) const {
        hipError_t error = hipFree(ptr);

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to deallocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount--;
    }

    void CudaInstance::syncHostDevice() const {
        hipError_t error = hipDeviceSynchronize();

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to synchronize host and device: "} + hipGetErrorString(error);
            RAISE_ERROR(DeviceError, message);
        }
    }

    void CudaInstance::queryDeviceCapabilities(cuBool_DeviceCaps &deviceCaps) const {
        const unsigned long long KiB = 1024;

        int device;
        hipError_t error = hipGetDevice(&device);

        if (error == hipSuccess) {
            hipDeviceProp_t deviceProp{};
            error = hipGetDeviceProperties(&deviceProp, device);

            if (error == hipSuccess) {
                strcpy(deviceCaps.name, deviceProp.name);
                deviceCaps.cudaSupported = true;
                deviceCaps.managedMem = mMemoryType == MemType::Managed;
                deviceCaps.minor = deviceProp.minor;
                deviceCaps.major = deviceProp.major;
                deviceCaps.warp = deviceProp.warpSize;
                deviceCaps.globalMemoryKiBs = deviceProp.totalGlobalMem / KiB;
                deviceCaps.sharedMemoryPerMultiProcKiBs = deviceProp.sharedMemPerMultiprocessor / KiB;
                deviceCaps.sharedMemoryPerBlockKiBs = deviceProp.sharedMemPerBlock / KiB;
            }
        }
    }

    CudaInstance::MemType CudaInstance::getMemoryType() const {
        return mMemoryType;
    }

    bool CudaInstance::isCudaDeviceSupported() {
        int device;
        hipError_t error = hipGetDevice(&device);
        return error == hipSuccess;
    }

    void CudaInstance::allocate(void* &ptr, size_t size) const {
        ptr = malloc(size);
        CHECK_RAISE_ERROR(ptr != nullptr, MemOpFailed, "Failed to allocate memory on the CPU");
        mHostAllocCount++;
    }

    void CudaInstance::deallocate(void* ptr) const {
        CHECK_RAISE_ERROR(ptr != nullptr, InvalidArgument, "Passed null ptr to free");
        free(ptr);
        mHostAllocCount--;
    }

    CudaInstance& CudaInstance::getInstanceRef() {
        CHECK_RAISE_ERROR(gInstance != nullptr, InvalidState, "No instance in the system");
        return (CudaInstance&) *gInstance;
    }

    CudaInstance* CudaInstance::getInstancePtr() {
        return (CudaInstance* ) gInstance;
    }

    bool CudaInstance::isInstancePresent() {
        return gInstance != nullptr;
    }

}
