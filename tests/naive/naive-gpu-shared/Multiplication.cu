#include "hip/hip_runtime.h"
// This file is part of the JetBrains-Research/CFPQ-on-GPGPU project.
// Project link https://github.com/JetBrains-Research/CFPQ-on-GPGPU
// Project is licensed under MIT license.
// License link https://github.com/JetBrains-Research/CFPQ-on-GPGPU/blob/master/LICENSE
//
// This source code files are used to compare the performance of the
// boolean matrix multiplication between two dense matrix multiplications implementations.

#include <iostream>
#include <naive-gpu-shared/Multiplication.h>
#include <naive-gpu-shared/Parameters.h>

#define cuda_handle_error(ans) { gpuAssert((ans), __FILE__, __LINE__); }

namespace naive_gpu_shared {

    using namespace gpu_lib;

    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
        if (code != hipSuccess) {
            std::cout << "GPU assert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
            if (abort) {
                exit(code);
            }
        }
    }

    __device__ bool matrix_was_changed;

    int gpu_lib::rows(int N) {
        return N / TYPE_SIZE + (N % TYPE_SIZE ? 1 : 0);
    }

    int gpu_lib::cols(int N) {
        return N;
    }

    inline size_t matrix_memsize(int N) {
        return rows(N) * cols(N) * sizeof(TYPE);
    }

    __device__ TYPE row_column_product(TYPE *A, TYPE *B, int cols) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int rows = cols / TYPE_SIZE + (cols % TYPE_SIZE ? 1 : 0);
        int row_start = blockIdx.y * cols;

        __shared__ TYPE A_shared[THREADS_PER_BLOCK];

        TYPE acc = 0;
        TYPE b_el;
        for (TYPE i = 0; i < rows; ++i) {
            if (i == (rows - 1) && x >= cols) {
                return 0;
            }
            if ((i % (THREADS_PER_BLOCK / TYPE_SIZE)) == 0) {
                A_shared[threadIdx.x] = A[row_start + i * TYPE_SIZE + threadIdx.x];
                if (THREADS_PER_BLOCK > 32) {
                    __syncthreads();
                }
            }

            __syncthreads();

            b_el = B[i * cols + x];
#pragma unroll
            for (TYPE b = 0; b < TYPE_SIZE; ++b) {
                if (b_el & 1) {
                    acc |= A_shared[(i % (THREADS_PER_BLOCK / TYPE_SIZE)) * TYPE_SIZE + b];
                }
                b_el >>= 1;
            }
        }

        return acc;
    }

    __device__ void or_value(TYPE *M, TYPE val) {
        TYPE old_value = *M;
        if (old_value != (val | old_value)) {
            matrix_was_changed = true;
            *M = val | old_value;
        }
    }

    __global__ void matrix_product_add(TYPE *A, TYPE *B, TYPE *C, int cols) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int row_start = blockIdx.y * cols;

        TYPE acc = row_column_product(A, B, cols);

        if (acc == 0) return;

        or_value(&C[row_start + x], acc);
    }

    __global__ void matrix_product(TYPE *A, TYPE *B, TYPE *C, int cols) {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int row_start = blockIdx.y * cols;

        TYPE acc = row_column_product(A, B, cols);

        if (acc == 0) return;

        C[row_start + x] = acc;
    }

    __global__ void matrix_add_to_left(TYPE *A, TYPE *B, int cols) {
        int index = blockIdx.y * cols + blockIdx.x * blockDim.x + threadIdx.x;

        if ((blockIdx.x * blockDim.x + threadIdx.x) >= cols) return;

        or_value(&A[index], B[index]);
    }

    void gpu_lib::synchronize() {
        hipDeviceSynchronize();
    }

    void set_value(int N, TYPE *d_M, int val) {
        cuda_handle_error(hipMemset(d_M, val, matrix_memsize(N)));
    }

    TYPE *gpu_lib::device_matrix_alloc(int N) {
        TYPE *d_M;
        cuda_handle_error(hipMalloc(reinterpret_cast<void **>(&d_M), matrix_memsize(N)));

        return d_M;
    }

    void gpu_lib::device_matrix_dealloc(TYPE *M) {
        cuda_handle_error(hipFree(M));
    }

    TYPE *gpu_lib::host_matrix_calloc(int N) {
        TYPE *M;
        cuda_handle_error(hipHostMalloc(reinterpret_cast<void **>(&M), matrix_memsize(N)));
        set_value(N, M, 0);
        return M;
    }

    void gpu_lib::host_matrix_dealloc(TYPE *M) {
        cuda_handle_error(hipHostFree(M));
    }

    void gpu_lib::gpu_to_cpu_transfer_async(int N, TYPE *d_M, TYPE *h_M) {
        cuda_handle_error(hipMemcpyAsync(h_M, d_M, matrix_memsize(N), hipMemcpyDeviceToHost));
    }

    void gpu_lib::cpu_to_gpu_transfer_async(int N, TYPE *h_M, TYPE *d_M) {
        cuda_handle_error(hipMemcpyAsync(d_M, h_M, matrix_memsize(N), hipMemcpyHostToDevice));
    }

    void set_flag() {
        bool flag = false;
        cuda_handle_error(hipMemcpyToSymbol(HIP_SYMBOL(matrix_was_changed), &flag, sizeof(bool)));
    }

    bool get_flag() {
        bool flag;
        cuda_handle_error(hipMemcpyFromSymbol(&flag, HIP_SYMBOL(matrix_was_changed), sizeof(bool)));

        return flag;
    }

    bool gpu_lib::matrix_product_add_wrapper(TYPE *A, TYPE *B, TYPE *C, int N, TYPE *tmp_matrix) {
        bool safe = (A == C) || (B == C);
        dim3 threads(THREADS_PER_BLOCK);
        dim3 blocks(cols(N) / THREADS_PER_BLOCK + (cols(N) % THREADS_PER_BLOCK ? 1 : 0), rows(N));

        set_flag();
        if (safe) {
            matrix_product <<<blocks, threads>>>(A, B, tmp_matrix, cols(N));
            synchronize();
            cuda_handle_error(hipGetLastError());
            matrix_add_to_left <<<blocks, threads>>>(C, tmp_matrix, cols(N));
            synchronize();
            cuda_handle_error(hipGetLastError());
        } else {
            matrix_product_add <<<blocks, threads>>>(A, B, C, cols(N));
            synchronize();
            cuda_handle_error(hipGetLastError());
        }

        return get_flag();
    }

}
