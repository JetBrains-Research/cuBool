/**********************************************************************************/
/*                                                                                */
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020 JetBrains-Research                                          */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/*                                                                                */
/**********************************************************************************/

#include <cubool/instance.hpp>
#include <cubool/matrix_dense.cuh>
#include <cubool/matrix_csr.cuh>
#include <cubool/details/error.hpp>
#include <string>
#include <cstring>

namespace cubool {

    Instance::~Instance() {
#ifdef CUBOOL_DEBUG
        // Notify user if he lost or forgot to release some library objects

        if (!mMatrixSet.empty()) {
            char message[2000];

            snprintf(message, 2000, "Some Matrix objects (%llu) were not properly deallocated by user",
                     (unsigned long long) mMatrixSet.size());

            sendMessage(CUBOOL_STATUS_INVALID_STATE, message);
        }

#endif //CUBOOL_DEBUG
        for (auto matrix: mMatrixSet) {
            matrix->~MatrixBase();
            deallocate(matrix);
        }
    }

    void Instance::createMatrixDense(MatrixDense *&matrix) {
        CuBoolCpuPtr_t mem = nullptr;
        allocate(&mem, sizeof(MatrixDense));

        matrix = new (mem) MatrixDense(*this);
        mMatrixSet.emplace(matrix);
    }

    void Instance::createMatrixCsr(MatrixCsr *&matrix) {
        CuBoolCpuPtr_t mem = nullptr;
        allocate(&mem, sizeof(MatrixCsr));

        matrix = new (mem) MatrixCsr(*this);
        mMatrixSet.emplace(matrix);
    }

    void Instance::validateMatrix(MatrixBase *matrix) {
        bool contains = mMatrixSet.find(matrix) != mMatrixSet.end();

        if (!contains)
            throw details::InvalidArgument("No such matrix for provided handler");
    }

    void Instance::destroyMatrix(MatrixBase* matrix) {
        auto itr = mMatrixSet.find(matrix);

        if (itr == mMatrixSet.end())
            throw details::InvalidArgument("No such matrix for provided handler");

        mMatrixSet.erase(itr);
        matrix->~MatrixBase();
        deallocate(matrix);
    }

    void Instance::allocateOnGpu(CuBoolGpuPtr_t* ptr, CuBoolSize_t size) const {
        if (!ptr)
            throw details::InvalidArgument("Passed null pointer to store allocated memory ptr");

        hipError_t error;

        switch (mMemoryType) {
            case CUBOOL_GPU_MEMORY_TYPE_GENERIC:
                error = hipMalloc(ptr, size);
                break;
            case CUBOOL_GPU_MEMORY_TYPE_MANAGED:
                error = hipMallocManaged(ptr, size);
                break;
            default:
                throw details::MemOpFailed("Failed to fined suitable allocator");
        }

        if (error != hipSuccess)
            throw details::MemOpFailed(std::string{"Failed to allocate Gpu memory: "} + hipGetErrorString(error));
    }

    void Instance::deallocateOnGpu(CuBoolGpuPtr_t ptr) const {
        if (!ptr)
            throw details::InvalidArgument("Passed null pointer to free");

        hipError_t error = hipFree(ptr);

        if (error != hipSuccess)
            throw details::MemOpFailed(std::string{"Failed to deallocate Gpu memory: "} + hipGetErrorString(error));
    }

    void Instance::syncHostDevice() const {
        hipError_t error = hipDeviceSynchronize();

        if (error != hipSuccess)
            throw details::MemOpFailed(std::string{"Failed to synchronize host and device: "} + hipGetErrorString(error));
    }

    bool Instance::isCudaDeviceSupported() {
        int device;
        hipError_t error = hipGetDevice(&device);
        return error == hipSuccess;
    }

    void Instance::queryDeviceCapabilities(CuBoolDeviceCaps &deviceCaps) {
        static const CuBoolSize_t KiB = 1024;

        int device;
        hipError_t error = hipGetDevice(&device);

        if (error == hipSuccess) {
            hipDeviceProp_t deviceProp{};
            error = hipGetDeviceProperties(&deviceProp, device);

            if (error == hipSuccess) {
                strcpy(deviceCaps.name, deviceProp.name);
                deviceCaps.cudaSupported = true;
                deviceCaps.minor = deviceProp.minor;
                deviceCaps.major = deviceProp.major;
                deviceCaps.warp = deviceProp.warpSize;
                deviceCaps.globalMemoryKiBs = deviceProp.totalGlobalMem / KiB;
                deviceCaps.sharedMemoryPerMultiProcKiBs = deviceProp.sharedMemPerMultiprocessor / KiB;
                deviceCaps.sharedMemoryPerBlockKiBs = deviceProp.sharedMemPerBlock / KiB;
            }
        }
    }

}
